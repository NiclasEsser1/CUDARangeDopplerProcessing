#include "CudaAlgorithm.cuh"
#include "CudaGPU.cuh"
#include "CudaBase.cuh"
#include "CudaVector.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>



CudaAlgorithm::CudaAlgorithm(CudaBase* obj_base, int width, int height, int depth, int c_depth)
{
    base = obj_base;
    device = base->getDevice();
	x_size = width;
    y_size = height;
	z_size = depth;
    color_depth = c_depth;
	allocated = false;
	floatBuffer = NULL;
	windowBuffer = NULL;
	complexBuffer = NULL;
}

CudaAlgorithm::~CudaAlgorithm()
{
	freeMemory();
}

void CudaAlgorithm::freeMemory()
{
	// If statement just prevents misleading output
	if(allocated)
	{
		printf("Free device memory\n");
		freeCudaVector(floatBuffer);
		freeCudaVector(windowBuffer);
		freeCudaVector(complexBuffer);
		freeCudaVector(charBuffer);
        allocated = false;
	}
}


bool CudaAlgorithm::initDeviceEnv()
{
	//Allocate device memory for processing chain
	total_required_mem = (x_size * y_size * sizeof(float)*2
        + x_size * y_size * sizeof(hipfftComplex)*2
        + x_size * y_size * color_depth * sizeof(unsigned char)
        + x_size * sizeof(float));

    printf("\nNeeded memory: %.2lf; free memory (%ld/%ld) MBytes\n",
        total_required_mem/(1024*1024),
        device->getFreeMemory()/(1024*1024),
        device->totalMemory()/(1024*1024)
    );
	if(device->checkMemory(total_required_mem))
	{
		floatBuffer = new CudaVector<float>(device, x_size * y_size, true);
		complexBuffer = new CudaVector<hipfftComplex>(device, x_size * y_size, true);
		charBuffer = new CudaVector<unsigned char>(device, x_size * y_size * color_depth, true);
		windowBuffer = new CudaVector<float>(device, x_size, true);
		allocated = true;
		return 1;
	}
	else
	{
        floatBuffer = NULL;
        complexBuffer = NULL;
        charBuffer = NULL;
        windowBuffer = NULL;
		printf("Not enough memory avaible on the used device, aborting... \n");
		return 0;
	}

}

void CudaAlgorithm::rangeDopplerAlgorithm(float* idata, char* odata, winType type, numKind kind, color_t colormap)
{
    CUDA_CHECK(hipMemcpy(floatBuffer->getDevPtr(), idata, x_size*y_size*sizeof(float), hipMemcpyHostToDevice));
    // floatBuffer->print();
    complexBuffer->resize((x_size/2+1) * y_size);
    charBuffer->resize((x_size/2+1) * y_size * color_depth);

    if(kind == COMPLEX)
    {
        base->setWindow(windowBuffer->getDevPtr(), x_size/2+1, type, kind);
        base->hilbertTransform(floatBuffer->getDevPtr(), complexBuffer->getDevPtr(), x_size, y_size);
        base->window(complexBuffer->getDevPtr(), windowBuffer->getDevPtr(), x_size/2+1, y_size);
        base->transpose(complexBuffer->getDevPtr(), x_size/2+1, y_size);
        base->window(complexBuffer->getDevPtr(), windowBuffer->getDevPtr(), y_size, x_size/2+1);
        base->transpose(complexBuffer->getDevPtr(), y_size, x_size/2+1);
        base->c2c1dFFT(complexBuffer->getDevPtr(), x_size/2+1, y_size);
    }
    else
    {
        base->setWindow(windowBuffer->getDevPtr(), x_size, type, kind);
        base->window(floatBuffer->getDevPtr(), windowBuffer->getDevPtr(), x_size, y_size);
        base->transpose(floatBuffer->getDevPtr(), x_size, y_size);
        base->window(floatBuffer->getDevPtr(), windowBuffer->getDevPtr(), y_size,x_size);
        base->transpose(floatBuffer->getDevPtr(), y_size, x_size);
        base->r2c1dFFT(complexBuffer->getDevPtr(), x_size, y_size, floatBuffer->getDevPtr());
    }
    // floatBuffer->save("floatbuffer.dat", x_size, y_size);
    floatBuffer->resize((x_size/2+1)*y_size);
    base->hermitianTranspose(complexBuffer->getDevPtr(), x_size/2+1, y_size);
    base->c2c1dFFT(complexBuffer->getDevPtr(), y_size, x_size/2+1);
    base->hermitianTranspose(complexBuffer->getDevPtr(), y_size, x_size/2+1);
    base->absolute(complexBuffer->getDevPtr(), floatBuffer->getDevPtr(), x_size/2+1, y_size);
    // floatBuffer->save(x_size/2+1, y_size);

    base->mapColors(floatBuffer->getDevPtr(), charBuffer->getDevPtr(), x_size/2+1, y_size, colormap);
    // charBuffer->save((x_size/2+1), y_size);

    CUDA_CHECK(hipMemcpy(odata, charBuffer->getDevPtr(), charBuffer->getSize(), hipMemcpyDeviceToHost));
}
