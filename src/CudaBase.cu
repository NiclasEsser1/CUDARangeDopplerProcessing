#include "hip/hip_runtime.h"
#include "CudaBase.cuh"

/**
_________
PUBLIC
_________
**/
CudaBase::CudaBase(CudaGPU* device)
{
	setDevice(device);
}

CudaBase::~CudaBase()
{

}



void CudaBase::setWindow(float* idata, int win_len, winType type)
{
	int tx = MAX_NOF_THREADS;
	int bx = win_len / MAX_NOF_THREADS + 1;

	dim3 blockSize(tx);
	dim3 gridSize(bx);

	switch (type)
	{
		case HAMMING:
			//printf("Calculate hamming window... ");
			windowHamming <<<gridSize, blockSize >>> (idata, win_len);
			// CUDA_CHECK(hipDeviceSynchronize());
			break;
		case HANN:
			//printf("Calculate hann window... ");
			windowHann <<<gridSize, blockSize >>> (idata, win_len);
			// CUDA_CHECK(hipDeviceSynchronize());
			break;
		case BARTLETT:
			//printf("Calculate bartlett window... ");
			windowBartlett <<<gridSize, blockSize >>> (idata, win_len);
			// CUDA_CHECK(hipDeviceSynchronize());
			break;
		case BLACKMAN:
			//printf("Calculate blackman window... ");
			windowBlackman <<<gridSize, blockSize >>> (idata, win_len);
			// CUDA_CHECK(hipDeviceSynchronize());
			break;
	}
	//printf("done!\n");
}

template <typename T>
void CudaBase::window(T* idata, float* window, int width, int height)
{
	int tx = MAX_NOF_THREADS;
	int bx = width/tx+1;
	int by = height;

	dim3 blockSize(tx);
	dim3 gridSize(bx, by);
	//printf("Performing windowing (real)... ");
	windowKernel<<<gridSize,blockSize>>>(idata, window, width, height);
	// CUDA_CHECK(hipDeviceSynchronize());
	//printf("done\n");
}
template void CudaBase::window<float>(float*, float*, int, int);
template void CudaBase::window<hipfftComplex>(hipfftComplex*, float*, int, int);

void CudaBase::absolute(hipfftComplex* idata, float* odata, int width, int height)
{
	int tx = MAX_NOF_THREADS;
	int bx = width/tx+1;
	int by = height;

	dim3 blockSize(tx);
	dim3 gridSize(bx, by);
	//printf("Calculating absolute values... ");
	absoluteKernel<<<gridSize,blockSize>>>(idata, odata, width, height);
	// CUDA_CHECK(hipDeviceSynchronize());
	//printf("done\n");
}

void CudaBase::hermitianTranspose(hipfftComplex* odata, int width, int height, hipfftComplex* idata)
{
	int tx = 32;
	int ty = 32;
	int bx = width/tx+1;
	int by = height/ty+1;
	dim3 blockSize(tx,ty);
	dim3 gridSize(bx,by);
	//printf("Transposing buffer... ");
	if(idata == NULL)
	{
		CudaVector<hipfftComplex>* temp = new CudaVector<hipfftComplex>(device, width*height);
		CUDA_CHECK(hipMemcpy(temp->getDevPtr(), odata, temp->getSize(), hipMemcpyDeviceToDevice));
		hermetianTransposeGlobalKernel<<<gridSize, blockSize>>>(temp->getDevPtr(), odata, width, height);
		temp->resize(0);
		delete(temp);
	}
	else
	{
		hermetianTransposeGlobalKernel<<<gridSize, blockSize>>>(idata, odata, width, height);
	}
	//CUDA_CHECK(hipDeviceSynchronize());
	//printf("done\n");
}

void CudaBase::fftshift(hipfftComplex* data, int n, int batch)
{
	int tx = MAX_NOF_THREADS;
	int bx = tx/n+1;
	int by = batch;
	dim3 blockSize(tx);
	dim3 gridSize(bx,by);
	if(batch > 1)
	{
		fftshift2d<<<gridSize, blockSize>>>(data, n, batch);
	}
	else
	{
		fftshift1d<<<gridSize, blockSize>>>(data, n);
	}
}

void CudaBase::encodeBmpToJpeg(unsigned char* idata, unsigned char* odata, int width, int height)
{
	nvjpegHandle_t nv_handle;
	nvjpegEncoderState_t nv_enc_state;
	nvjpegEncoderParams_t nv_enc_params;
	hipStream_t stream;
	nvjpegImage_t source;

	source.channel[0] = idata;
	source.pitch[0] = width*3;

	CUDA_CHECK(hipStreamCreate(&stream));

	// initialize nvjpeg structures
	CUDA_CHECK_NVJPEG(nvjpegCreateSimple(&nv_handle));
	CUDA_CHECK_NVJPEG(nvjpegEncoderStateCreate(nv_handle, &nv_enc_state, stream));
	CUDA_CHECK_NVJPEG(nvjpegEncoderParamsCreate(nv_handle, &nv_enc_params, stream));
	//CUDA_CHECK_NVJPEG(nvjpegEncoderParamsSetSamplingFactors(nv_enc_params, NVJPEG_CSS_444, stream));

	// Compress image
	CUDA_CHECK_NVJPEG(nvjpegEncodeImage(nv_handle, nv_enc_state, nv_enc_params, &source, NVJPEG_INPUT_RGB, width, height, stream));
	CUDA_CHECK(hipStreamSynchronize(stream));
	CUDA_CHECK(hipDeviceSynchronize());

	// get compressed stream size
	size_t length;
	CUDA_CHECK_NVJPEG(nvjpegEncodeRetrieveBitstream(nv_handle, nv_enc_state, NULL, &length, stream));


	CUDA_CHECK(hipStreamSynchronize(stream));
	CUDA_CHECK_NVJPEG(nvjpegEncodeRetrieveBitstream(nv_handle, nv_enc_state, odata, &length, 0));
	printf("Length is: %ld \n", length);
	CUDA_CHECK(hipDeviceSynchronize());
}

template <typename T>
void CudaBase::transpose(T* odata, int width, int height, T* idata)
{
	int tx = 32;
	int ty = 32;
	int bx = width/tx+1;
	int by = height/ty+1;
	dim3 blockSize(tx,ty);
	dim3 gridSize(bx,by);
	//printf("Transposing buffer... ");
	if(idata == NULL)
	{
		CudaVector<T>* temp = new CudaVector<T>(device, width*height);
		CUDA_CHECK(hipMemcpy(temp->getDevPtr(), odata, temp->getSize(), hipMemcpyDeviceToDevice));
		transposeGlobalKernel<<<gridSize, blockSize>>>(temp->getDevPtr(), odata, width, height);
		temp->resize(0);
		delete(temp);
	}
	else
	{
		transposeGlobalKernel<<<gridSize, blockSize>>>(idata, odata, width, height);
	}
	//CUDA_CHECK(hipDeviceSynchronize());
	//printf("done\n");
}
template void CudaBase::transpose<float>(float*, int, int, float*);
template void CudaBase::transpose<hipfftComplex>(hipfftComplex*, int, int, hipfftComplex*);

template <typename T>
void CudaBase::transposeShared(T* odata, int width, int height, T* idata)
{
	int tx = 32;
	int ty = 32;
	int bx = width/tx+1;
	int by = height/ty+1;
	dim3 blockSize(tx,ty);
	dim3 gridSize(bx,by);
	//printf("Transposing buffer... ");
	if(idata == NULL)
	{
		CudaVector<T>* temp = new CudaVector<T>(device, width*height);
		CUDA_CHECK(hipMemcpy(temp->getDevPtr(), odata, temp->getSize(), hipMemcpyDeviceToDevice));
		transposeSharedKernel<<<gridSize, blockSize>>>(temp->getDevPtr(), odata, height);
		temp->resize(0);
		delete(temp);
	}
	else
	{
		transposeSharedKernel<<<gridSize, blockSize>>>(idata, odata, height);
	}
	//CUDA_CHECK(hipDeviceSynchronize());
	//printf("done\n");
}
template void CudaBase::transposeShared<float>(float*, int, int, float*);
template void CudaBase::transposeShared<hipfftComplex>(hipfftComplex*, int, int, hipfftComplex*);


template <typename T>
T CudaBase::max(T* idata, int width, int height)
{
	int count = width*height;
	int tx = MAX_NOF_THREADS;
	int bx = count/tx;
	float max_val = 0;
	dim3 blockSize(tx);
	dim3 gridSize(bx);

	CudaVector<T>* temp = new CudaVector<T>(device, count);
	CUDA_CHECK(hipMemcpy(temp->getDevPtr(), idata, temp->getSize(), hipMemcpyDeviceToDevice));

	maxKernel<T><<<gridSize, blockSize>>>(temp->getDevPtr(), count);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(&max_val, temp->getDevPtr(0), sizeof(T), hipMemcpyDeviceToHost));
	temp->resize(0);
	delete(temp);
	return max_val;
}
template float CudaBase::max<float>(float*, int, int);
template int CudaBase::max<int>(int*, int, int);
template char CudaBase::max<char>(char*, int, int);
template double CudaBase::max<double>(double*, int, int);



template <typename T>
T CudaBase::min(T* idata, int width, int height)
{
	int count = width*height;
	int tx = MAX_NOF_THREADS;
	int bx = count/tx;
	float min_val = 0;
	dim3 blockSize(tx);
	dim3 gridSize(bx);

	CudaVector<T>* temp = new CudaVector<T>(device, count);
	CUDA_CHECK(hipMemcpy(temp->getDevPtr(), idata, temp->getSize(), hipMemcpyDeviceToDevice));

	minKernel<T><<<gridSize, blockSize>>>(temp->getDevPtr(), count);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(&min_val, temp->getDevPtr(0), sizeof(T), hipMemcpyDeviceToHost));
	temp->resize(0);
	delete(temp);
	return min_val;
}
template float CudaBase::min<float>(float*, int, int);
template int CudaBase::min<int>(int*, int, int);
template char CudaBase::min<char>(char*, int, int);
template double CudaBase::min<double>(double*, int, int);



void CudaBase::mapColors(float* idata, unsigned char* odata, int width, int height, color_t type)
{
	int tx = 32;
	int ty = 32;
	int bx = width/tx+1;
	int by = height/ty+1;
	CUDA_CHECK(hipDeviceSynchronize());
	float max_v = max(idata, width, height);
	float min_v = min(idata, width, height);
	dim3 blockSize(tx,ty);
	dim3 gridSize(bx,by);

	switch(type)
	{
		case JET:
			colormapJet<<<gridSize,blockSize>>>(idata, odata, max_v, min_v, width, height);
			// CUDA_CHECK(hipDeviceSynchronize());
			break;
		case VIRIDIS:
			colormapViridis<<<gridSize,blockSize>>>(idata, odata, max_v, min_v, width, height);
			// CUDA_CHECK(hipDeviceSynchronize());
		case ACCENT:
			colormapAccent<<<gridSize,blockSize>>>(idata, odata, max_v, min_v, width, height);
			// CUDA_CHECK(hipDeviceSynchronize());
		case MAGMA:
			colormapMagma<<<gridSize,blockSize>>>(idata, odata, max_v, min_v, width, height);
			// CUDA_CHECK(hipDeviceSynchronize());
			break;
		case INFERNO:
			colormapInferno<<<gridSize,blockSize>>>(idata, odata, max_v, min_v, width, height);
			// CUDA_CHECK(hipDeviceSynchronize());
			break;
		case BLUE:
			colormapBlue<<<gridSize,blockSize>>>(idata, odata, max_v, min_v, width, height);
			// CUDA_CHECK(hipDeviceSynchronize());
			break;
	}
}


/*
*	FFT functions
*/

void CudaBase::r2c1dFFT(hipfftComplex* odata, int n, int batch, hipfftReal* idata)
{
	//printf("Performing 1D FFT (r2c)... ");
	hipfftHandle plan;
	// Plan for FFT
	CUDA_CHECK_FFT(hipfftPlan1d(&plan, n, HIPFFT_R2C, batch));
	if(idata == NULL)
	{
		CUDA_CHECK_FFT(hipfftExecR2C(plan, (hipfftReal*)odata, odata));
	}
	else
	{
		CUDA_CHECK_FFT(hipfftExecR2C(plan, idata, odata));
	}
	CUDA_CHECK_FFT(hipfftDestroy(plan));
	//printf("done! \n");
}

void CudaBase::c2c1dInverseFFT(hipfftComplex* idata, int n, int batch)
{
	//printf("Performing 1D inverse FFT (c2c)... ");
	hipfftHandle plan;
	// Plan for FFT
	CUDA_CHECK_FFT(hipfftPlan1d(&plan, n, HIPFFT_C2C, batch));
	CUDA_CHECK_FFT(hipfftExecC2C(plan, idata, idata, HIPFFT_BACKWARD));
	CUDA_CHECK_FFT(hipfftDestroy(plan));
	//printf("done! \n");
}

void CudaBase::c2c1dFFT(hipfftComplex* idata, int n, int batch)
{
	//printf("Performing 1D FFT (c2c)... ");
	hipfftHandle plan;
	// Plan for FFT
	CUDA_CHECK_FFT(hipfftPlan1d(&plan, n, HIPFFT_C2C, batch));
	// Execute in place FFT (destination = source)
	CUDA_CHECK_FFT(hipfftExecC2C(plan, idata, idata, HIPFFT_FORWARD));
	CUDA_CHECK_FFT(hipfftDestroy(plan));
	//printf("done! \n");
}

void CudaBase::r2cManyFFT(float* idata, hipfftComplex *odata, int *nfft, int  rank)
{
	hipfftHandle plan;
	int length = nfft[0];
	// Plan for FFT
	CUDA_CHECK_FFT(hipfftPlanMany(
		&plan, rank, nfft,
		nfft, length, rank,
		nfft, length, rank,
		HIPFFT_R2C, length
	));
	// Execute in place FFT (destination = source)
	CUDA_CHECK_FFT(hipfftExecR2C(plan, idata, odata));
	CUDA_CHECK_FFT(hipfftDestroy(plan));
}

void CudaBase::hilbertTransform(float* idata, hipfftComplex* odata, int n, int batch)
{
	//printf("Performing hilbert transform... \n");
	r2c1dFFT(odata, n, batch, idata);
	c2c1dInverseFFT(odata, n/2+1, batch);
	//printf("done\n");
}



void CudaBase::printWindowTaps(float* idata, int win_len)
{
	float* help = (float*)malloc(win_len*sizeof(float));
	CUDA_CHECK(hipMemcpy(help, idata, win_len*sizeof(float), hipMemcpyDeviceToHost));
	for(int i = 0; i < win_len; i++)
		printf("Tap[%d] = %f\n",i,help[i]);
}
