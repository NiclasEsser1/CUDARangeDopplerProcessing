#include "CudaBase.cuh"
#include "CudaGPU.cuh"
#include "CudaKernels.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <unistd.h>
#include <iostream>

/**
_________
PUBLIC
_________
**/
CudaBase::CudaBase(CudaGPU* device)
{
	setDevice(device);
	setLength(0);
	setHeight(0);
}

CudaBase::~CudaBase()
{
	freeMemory();
}

void CudaBase::freeMemory()
{
	freeCudaVector(floatBuffer);
	freeCudaVector(windowBuf);
	freeCudaVector(complexBuffer);
	freeCudaVector(transposeBuf);
}

bool CudaBase::initDeviceEnv()
{
	//Allocate device memory for processing chain
	printf("Allocate memory for processing buffer\n");
	total_used_mem = x_size * y_size * 3  + x_size;
	printf("needed mem: %ld and total avaible mem: %ld\n", total_used_mem/(1024*1024),device->totalMemory()/(1024*1024));
	if(total_used_mem < device->totalMemory())
	{
		floatBuffer = new CudaVector<float>(device, x_size * y_size);
		complexBuffer = new CudaVector<hipfftComplex>(device, x_size * y_size);
		transposeBuf = new CudaVector<hipfftComplex>(device, x_size * y_size);
		windowBuf = new CudaVector<float>(device, x_size);
		return 1;
	}
	else
	{
		printf("Not enoguh memory avaible on the used device, aborting... \n");
		return 0;
	}
}

void CudaBase::setWindow(winType type, numKind kind)
{
	win_type = type;
	win_kind = kind;
	win_len = x_size;
	calculateWindowTaps();
}

void CudaBase::windowReal(float* idata, int width, int height)
{
	int tx = MAX_NOF_THREADS;
	int bx = width/tx+1;
	int by = height;

	dim3 blockSize(tx);
	dim3 gridSize(bx, by);

	//printf("Performing windowing (real)... ");
	windowMultiplyReal<<<gridSize,blockSize>>>((float*)idata, windowBuf->getDevPtr(), width, height);
	//printf("done\n");
}

void CudaBase::windowCplx(hipfftComplex* idata, int width, int height)
{
	int tx = MAX_NOF_THREADS;
	int bx = width/tx+1;
	int by = height;

	dim3 blockSize(tx);
	dim3 gridSize(bx, by);

	//printf("Performing windowing (complex)... ");
	windowMultiplyCplx<<<gridSize,blockSize>>>(idata, windowBuf->getDevPtr(), width, height);
	//printf("done\n");
}

void CudaBase::calculateWindowTaps()
{
	int tx = MAX_NOF_THREADS;
	int bx = win_len / MAX_NOF_THREADS + 1;

	dim3 blockSize(tx);
	dim3 gridSize(bx);

	switch (win_type)
	{
		case HAMMING:
			//printf("Calculate hamming window... ");
			windowHamming <<<gridSize, blockSize >>> (windowBuf->getDevPtr(), win_len);
			CUDA_CHECK(hipDeviceSynchronize());
			break;
		case HANN:
			//printf("Calculate hann window... ");
			windowHann <<<gridSize, blockSize >>> (windowBuf->getDevPtr(), win_len);
			CUDA_CHECK(hipDeviceSynchronize());
			break;
		case BARTLETT:
			//printf("Calculate bartlett window... ");
			windowBartlett <<<gridSize, blockSize >>> (windowBuf->getDevPtr(), win_len);
			CUDA_CHECK(hipDeviceSynchronize());
			break;
		case BLACKMAN:
			//printf("Calculate blackman window... ");
			windowBlackman <<<gridSize, blockSize >>> (windowBuf->getDevPtr(), win_len);
			CUDA_CHECK(hipDeviceSynchronize());
			break;
	}
	//printf("done!\n");
}

void CudaBase::absolute(hipfftComplex* idata, float* odata, int width, int height)
{
	int tx = MAX_NOF_THREADS;
	int bx = width/tx+1;
	int by = height;

	dim3 blockSize(tx);
	dim3 gridSize(bx, by);
	//printf("Calculating absolute values... ");
	absoluteKernel<<<gridSize,blockSize>>>(idata, odata, width, height);
	//printf("done\n");
}

void CudaBase::transpose(hipfftComplex* idata, hipfftComplex* odata, int width, int height)
{
	int tx = 32;
	int ty = 32;
	int bx = width/tx+1;
	int by = height/ty+1;
	dim3 blockSize(tx,ty);
	dim3 gridSize(bx,by);
	//printf("Transposing buffer... ");
	transposeBufferGlobalCplx<<<gridSize, blockSize>>>(idata, odata, width, height);
	//printf("done\n");
}

void CudaBase::r2c1dFFT(float* idata, hipfftComplex *odata, int n, int batch)
{
	//printf("Performing 1D FFT (r2c)... ");
	hipfftHandle plan;
	// Plan for FFT
	CUDA_CHECK_FFT(hipfftPlan1d(&plan, n, HIPFFT_R2C, batch));
	CUDA_CHECK_FFT(hipfftExecR2C(plan, (hipfftReal*)idata, odata));
	CUDA_CHECK_FFT(hipfftDestroy(plan));
	//printf("done! \n");
}

void CudaBase::c2c1dIFFT(hipfftComplex* idata, int n, int batch)
{
	//printf("Performing 1D inverse FFT (c2c)... ");
	hipfftHandle plan;
	// Plan for FFT
	CUDA_CHECK_FFT(hipfftPlan1d(&plan, n, HIPFFT_C2C, batch));
	CUDA_CHECK_FFT(hipfftExecC2C(plan, idata, idata, HIPFFT_BACKWARD));
	CUDA_CHECK_FFT(hipfftDestroy(plan));
	//printf("done! \n");
}

void CudaBase::c2c1dFFT(hipfftComplex* idata, int n, int batch)
{
	//printf("Performing 1D FFT (c2c)... ");
	hipfftHandle plan;
	// Plan for FFT
	CUDA_CHECK_FFT(hipfftPlan1d(&plan, n, HIPFFT_C2C, batch));
	// Execute in place FFT (destination = source)
	CUDA_CHECK_FFT(hipfftExecC2C(plan, idata, idata, HIPFFT_FORWARD));
	CUDA_CHECK_FFT(hipfftDestroy(plan));
	//printf("done! \n");
}

void CudaBase::r2cManyFFT(float* idata, hipfftComplex *odata, int *nfft, int  rank)
{
	hipfftHandle plan;
	int length = nfft[0];
	// Plan for FFT
	CUDA_CHECK_FFT(hipfftPlanMany(
		&plan, rank, nfft,
		nfft, length, rank,
		nfft, length, rank,
		HIPFFT_R2C, length
	));
	// Execute in place FFT (destination = source)
	CUDA_CHECK_FFT(hipfftExecR2C(plan, idata, odata));
	CUDA_CHECK_FFT(hipfftDestroy(plan));
}

void CudaBase::hilbertTransform(float* idata, hipfftComplex* odata, int n, int batch)
{
	//printf("Performing hilbert transform... \n");
	r2c1dFFT(idata, odata, n, batch);
	c2c1dIFFT(odata, n/2+1, batch);
	//printf("done\n");
}

void CudaBase::printWindowTaps()
{
	float* help = (float*)malloc(win_len*sizeof(float));
	CUDA_CHECK(hipMemcpy(help, windowBuf->getDevPtr(), win_len*sizeof(float), hipMemcpyDeviceToHost));
	for(int i = 0; i < win_len; i++)
		printf("Tap[%d] = %f\n",i,help[i]);
}
