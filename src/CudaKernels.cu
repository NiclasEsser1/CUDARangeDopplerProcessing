#include "hip/hip_runtime.h"
#include "CudaKernels.cuh"



__global__ void windowHamming(float* idata, int length)
{
	int tidx = threadIdx.x + blockIdx.x*blockDim.x;
	if (tidx < length)
	{
		idata[tidx] = 0.54 - 0.46 * cos(2*tidx*PI_F / (length - 1));
	}
}
__global__ void windowHann(float* idata, int length)
{
	int tidx = threadIdx.x + blockIdx.x*blockDim.x;
	if (tidx < length)
	{
		idata[tidx] = 0.5*(1 + cos(2*tidx*PI_F / (length - 1)));
	}
}
__global__ void windowBartlett(float* idata, int length)
{
	int tidx = threadIdx.x + blockIdx.x*blockDim.x;
	if (tidx < length)
	{
		idata[tidx] = 0;
	}
}
__global__ void windowBlackman(float* idata, int length)
{
	int tidx = threadIdx.x + blockIdx.x*blockDim.x;
	if (tidx < length)
	{
		idata[tidx] = 0.74 / 2 * -0.5 * cos(2 * PI_F*tidx / (length - 1)) + 0.16 / 2 * sin(4 * PI_F*tidx / (length - 1));
	}
}

__global__ void windowKernel(float* idata, float* window, int width, int height)
{
	int tidx = threadIdx.x + blockIdx.x*blockDim.x;
	int tidy = threadIdx.y + blockIdx.y*blockDim.y;
	if(tidx < width && tidy < height)
	{
		idata[tidy * width + tidx] = window[tidx] * idata[tidy * width + tidx];
	}
}


__global__ void windowKernel(hipfftComplex* idata, float* window, int width, int height)
{
	int tidx = threadIdx.x + blockIdx.x*blockDim.x;
	int tidy = threadIdx.y + blockIdx.y*blockDim.y;
	if(tidx < width && tidy < height)
	{
		idata[tidy * width + tidx].x = window[tidx] * idata[tidy * width + tidx].x;
		idata[tidy * width + tidx].y = window[tidx] * idata[tidy * width + tidx].y;
	}
}

__global__ void transposeGlobalKernel(float* idata, float* odata, int width, int height)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y+ threadIdx.y;

	if(tidx < width && tidy < height)
	{
		odata[tidx*height + tidy] = idata[tidy*width + tidx];
	}
}

__global__ void hermetianTransposeGlobalKernel(hipfftComplex* idata, hipfftComplex* odata, int width, int height)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	if(tidx < width && tidy < height)
	{
		odata[tidx*height + tidy].x = idata[tidy*width + tidx].x;
		odata[tidx*height + tidy].y = (-1)*idata[tidy*width + tidx].y;
	}
}


__global__ void transposeGlobalKernel(hipfftComplex* idata, hipfftComplex* odata, int width, int height)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;
	if(tidx < width && tidy < height)
	{
		odata[tidx*height + tidy].x = idata[tidy*width + tidx].x;
		odata[tidx*height + tidy].y = idata[tidy*width + tidx].y;
	}
}

__global__ void transposeSharedKernel(float* idata, float* odata, int height)
{
	__shared__ float tile[32][32];

	int x = blockIdx.x * 32 + threadIdx.x;
	int y = blockIdx.y * 32 + threadIdx.y;
	int width = gridDim.x * 32;

	for (int j = 0; j < 32; j += height/32)
		tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

	__syncthreads();

	x = blockIdx.y * 32 + threadIdx.x;  // transpose block offset
	y = blockIdx.x * 32 + threadIdx.y;

	for (int j = 0; j < 32; j += height/32)
		odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

__global__ void transposeSharedKernel(hipfftComplex* idata, hipfftComplex* odata, int height)
{
	__shared__ hipfftComplex tile[32][32];

	  int x = blockIdx.x * 32 + threadIdx.x;
	  int y = blockIdx.y * 32 + threadIdx.y;
	  int width = gridDim.x * 32;

	  for (int j = 0; j < 32; j += height/32)
	  {
		  tile[threadIdx.y+j][threadIdx.x].x = idata[(y+j)*width + x].x;
		  tile[threadIdx.y+j][threadIdx.x].y = idata[(y+j)*width + x].y;
	  }

	  __syncthreads();

	  x = blockIdx.y * 32 + threadIdx.x;  // transpose block offset
	  y = blockIdx.x * 32 + threadIdx.y;

	  for (int j = 0; j < 32; j += height/32)
	  {
		  odata[(y+j)*width + x].x = tile[threadIdx.x][threadIdx.y + j].x;
		  odata[(y+j)*width + x].y = tile[threadIdx.x][threadIdx.y + j].y;
	  }
}

__global__ void absoluteKernel(hipfftComplex* idata, float* odata, int width, int height)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	if(tidx < width && tidy < height)
	{
		odata[tidy*width + tidx] = sqrt(idata[tidy*width + tidx].x * idata[tidy*width + tidx].x + idata[tidy*width + tidx].y*idata[tidy*width + tidx].y);
	}
}



__global__ void colormapJet(float* idata, unsigned char* odata, float max, float min, int width, int height)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;
	int colormap_index = (int)((idata[tidx + width*tidy]-min)/(max-min)*(JET_SIZE-1));

	// if(tidx == 20 || tidx == 30 || tidx == 50)
	// 	printf("Index: %d, red: %d, blue: %d, green: %d, max: %f, min: %f; Value: %f position[%d][%d]\n",
	// 		colormap_index,  (unsigned)colormap_blue[colormap_index][0],
	// 		(unsigned)colormap_blue[colormap_index][1],(unsigned)colormap_blue[colormap_index][2],
	// 		max, min, idata[tidx + width*tidy],
	// 		tidy, tidx);

	if(tidx < width && tidy < height)
	{
		odata[(tidx + width * tidy) * 3 + 0] = (unsigned char)255*colormap_jet[colormap_index][0];
		odata[(tidx + width * tidy) * 3 + 1] = (unsigned char)255*colormap_jet[colormap_index][1];
		odata[(tidx + width * tidy) * 3 + 2] = (unsigned char)255*colormap_jet[colormap_index][2];
	}
}

__global__ void colormapViridis(float* idata, unsigned char* odata, float max, float min, int width, int height)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;
	int colormap_index = (int)(idata[tidx + width*tidy]-min)/(max-min)*(VIRIDIS_SIZE-1);
	if(tidx < width && tidy < height)
	{
		odata[(tidx + width*tidy) * 3 + 0] = (unsigned char)255*colormap_viridis[colormap_index][0];
		odata[(tidx + width*tidy) * 3 + 1] = (unsigned char)255*colormap_viridis[colormap_index][1];
		odata[(tidx + width*tidy) * 3 + 2] = (unsigned char)255*colormap_viridis[colormap_index][2];
	}
}


__global__ void colormapAccent(float* idata, unsigned char* odata, float max, float min, int width, int height)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;
	int colormap_index = (int)(idata[tidx + width*tidy]-min)/(max-min)*(ACCENT_SIZE-1);
	if(tidx < width && tidy < height)
	{
		odata[(tidx + width*tidy) * 3 + 0] = colormap_accent[colormap_index][0];
		odata[(tidx + width*tidy) * 3 + 1] = colormap_accent[colormap_index][1];
		odata[(tidx + width*tidy) * 3 + 2] = colormap_accent[colormap_index][2];
	}
}

__global__ void colormapMagma(float* idata, unsigned char* odata, float max, float min, int width, int height)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;
	int colormap_index = (int)((idata[tidx + width*tidy]-min)/(max-min)*(MAGMA_SIZE-1));
	if(tidx < width && tidy < height)
	{
		odata[(tidx + width*tidy) * 3 + 0] = (unsigned char)255*colormap_magma[colormap_index][0];
		odata[(tidx + width*tidy) * 3 + 1] = (unsigned char)255*colormap_magma[colormap_index][1];
		odata[(tidx + width*tidy) * 3 + 2] = (unsigned char)255*colormap_magma[colormap_index][2];
	}
}

__global__ void colormapInferno(float* idata, unsigned char* odata, float max, float min, int width, int height)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;
	int colormap_index = (int)(idata[tidx + width*tidy]-min)/(max-min)*(INFERNO_SIZE-1);
	if(tidx < width && tidy < height)
	{
		odata[(tidx + width*tidy) * 3 + 0] = (unsigned char)255*colormap_inferno[colormap_index][0];
		odata[(tidx + width*tidy) * 3 + 1] = (unsigned char)255*colormap_inferno[colormap_index][1];
		odata[(tidx + width*tidy) * 3 + 2] = (unsigned char)255*colormap_inferno[colormap_index][2];
	}
}

__global__ void colormapBlue(float* idata, unsigned char* odata, float max, float min, int width, int height)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;
	int colormap_index = (int)(idata[tidx + width*tidy]-min)/(max-min)*(BLUE_SIZE-1);
	if(tidx < width && tidy < height)
	{
		odata[(tidx + width*tidy) * 3 + 0] = colormap_blue[colormap_index][0];
		odata[(tidx + width*tidy) * 3 + 1] = colormap_blue[colormap_index][1];
		odata[(tidx + width*tidy) * 3 + 2] = colormap_blue[colormap_index][2];
	}
}

template <typename T>__global__ void maxKernel(T* idata, int count)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int i = count/2;

	if(tidx < i)
	{
		while(i != 0)
		{
			if(idata[tidx] < idata[tidx+i])
				idata[tidx] = idata[tidx+i];
			__syncthreads();
			if(idata[0] < idata[tidx])
				idata[0] = idata[tidx];
			__syncthreads();
			i /= 2;
		}
	}
}
template __global__ void maxKernel<float>(float*, int);
template __global__ void maxKernel<int>(int*, int);
template __global__ void maxKernel<char>(char*, int);
template __global__ void maxKernel<double>(double*, int);

template <typename T>__global__ void minKernel(T* idata, int count)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int i = count/2;

	if(tidx < i)
	{
		while(i != 0)
		{
			if(idata[tidx] > idata[tidx+i])
				idata[tidx] = idata[tidx+i];
			__syncthreads();
			if(idata[0] > idata[tidx])
				idata[0] = idata[tidx+i];
			__syncthreads();
			i /= 2;
		}
	}
}
template __global__ void minKernel<float>(float*, int);
template __global__ void minKernel<int>(int*, int);
template __global__ void minKernel<char>(char*, int);
template __global__ void minKernel<double>(double*, int);
