#include "cudagpu.cuh"


CudaGPU::CudaGPU(int devNum)
{
    id = devNum;
    printf("Starting CUDA device query...\n");
    int deviceCount = 0;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    CUDA_CHECK(hipSetDevice(id));
    CUDA_CHECK(hipGetDeviceProperties(&prop, id));
    if (deviceCount == 0)
    {
        printf("There are no available device(s) that support CUDA\n");
    }
    else
    {
        printf("Detected %d CUDA Capable device(s), choosed device %d\n", deviceCount, id);
    }
    (hipMemGetInfo(&free_mem, &total_mem));
}

CudaGPU::~CudaGPU()
{
    hipDeviceReset();
}

void CudaGPU::setDeviceID(int val)
{
    id = val;
}

int CudaGPU::getDeviceID()
{
    return id;
}

hipDeviceProp_t CudaGPU::getProperties()
{
    return prop;
}
int CudaGPU::checkMemory(size_t size, bool print)
{
    if(size != 0 && print)
		printf("GPU free mem: (%.2f/%.2f) MBytes\n", (float)free_mem/(1024*1024), (float)total_mem/(1024*1024));
	if(free_mem < size)
		return 0;
    free_mem = free_mem - size;
	return 1;
}
