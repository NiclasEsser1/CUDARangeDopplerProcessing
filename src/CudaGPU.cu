#include "CudaGPU.cuh"


CudaGPU::CudaGPU(int devNum)
{
    printf("Starting CUDA device query...\n");
    int deviceCount = 0;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    checkMemory();
    setDeviceID(devNum);
    CUDA_CHECK(hipSetDevice(id));
    CUDA_CHECK(hipGetDeviceProperties(&prop, id));
    if (deviceCount == 0)
    {
        printf("There are no available device(s) that support CUDA\n");
    }
    else
    {
        printf("Detected %d CUDA Capable device(s), choosed device %d\n", deviceCount, id);
    }
}

CudaGPU::~CudaGPU()
{
    hipDeviceReset();
}

void CudaGPU::setDeviceID(int val)
{
    id = val;
}

int CudaGPU::getDeviceID()
{
    return id;
}

hipDeviceProp_t CudaGPU::getProperties()
{
    hipGetDeviceProperties(&prop, id);
    return prop;
}
int CudaGPU::checkMemory(size_t size, bool print)
{
    hipMemGetInfo(&free_mem, &total_mem);
    if(size != 0 && print)
        printf("GPU free mem: (%.2f/%.2f) MBytes\n", (float)free_mem/(1024*1024), (float)total_mem/(1024*1024));
    if(free_mem < size)
        return 0;
    return 1;
}
